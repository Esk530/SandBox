#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/async/copy.h>
#include <thrust/async/transform.h>
#include <thrust/async/for_each.h>

#include "poTimer.h"

__global__ void vector_add(thrust::device_ptr<float> A, thrust::device_ptr<float> B, thrust::device_ptr<float> C , int N)
{
    int id = threadIdx.x + blockDim.x *blockIdx.x;
    if(id >= N)return;
    C[id] = A[id] + B[id];
}

int main(int argc, char** argv)
{
    int N = 10000000;
    thrust::device_vector<float> A(N,10.0f);
    std::cout << A[0] << std::endl;
    thrust::device_vector<float> B(N,20.0f);
    thrust::device_vector<float> C(N);

    std::vector<float> hA(N,10.0f);
    std::vector<float> hB(N,20.0f);
    std::vector<float> hC(N);

    auto a = A.data();
    auto b = B.data();
    auto c = C.data();
    {
        po::simpleTimer timer("for_each");
        for(int i = 0; i < 1000 ;i++)
        {
            thrust::for_each(thrust::counting_iterator<int>(0),thrust::counting_iterator<int>(N),[a,b,c]__device__(int id)
            {
                c[id] = a[id] + b[id];
                // C[id] = A[id] + B[id];
            }
            );
        }
        std::cout << "Out" << std::endl;

    }
    thrust::device_event e(thrust::new_stream);
    {
        po::simpleTimer timer("for_each_async");

        for(int i = 0; i < 1000; i++)
        {
            e = thrust::async::for_each(
                thrust::device.after(e),thrust::counting_iterator<int>(0),thrust::counting_iterator<int>(N),
                [a,b,c]__device__(int id){
                    c[id] = a[id] + b[id];
                }
            );
        }
        e.wait();
        std::cout << "Out" << std::endl;
    }

    {
        po::simpleTimer timer("global");
        int nThread = 512;
        int nBlocks = N/nThread + 1;
        vector_add<<<nBlocks,nThread>>>(a,b,c,N);
        std::cout << "Out" << std::endl;
        hipDeviceSynchronize();
    }

    {
        po::simpleTimer timer("CPU 1core");
            for(int i = 0; i < N;i++)
            {
                hC[i] = hA[i] + hB[i];
            }
    }

}